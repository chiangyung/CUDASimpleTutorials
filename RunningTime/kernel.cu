#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//CUDA RunTime API
#include <hip/hip_runtime.h>

#define DATA_SIZE 1000

int data[DATA_SIZE];

void GenerateNumbers(int *number, int size)
{
	for (int i = 0; i < size; i++) {
		number[i] = rand() % 10;
	}
}

//CUDA 初始化
bool InitCUDA()
{
	int count;

	//取得支持Cuda的装置的数目
	hipGetDeviceCount(&count);

	if (count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;

	for (i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if (prop.major >= 1) {
				break;
			}
		}
	}

	if (i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

__global__ static void sumOfSquares(int *num, int *result, clock_t *time)
{
	int sum = 0;
	clock_t start = clock();
	for (int i = 0; i < DATA_SIZE; i++) {
		sum += num[i] * num[i];
	}
	*result = sum;
	*time = clock() - start;
}

int main()
{
	//CUDA 初始化
	if (!InitCUDA()) {
		return 0;
	}

	//生成随机数
	GenerateNumbers(data, DATA_SIZE);

	/*把数据复制到显卡内存中*/

	int *gpudata, *result;
	clock_t *time;

	//cudaMalloc 取得一块显卡内存 ( 其中result用来存储计算结果 )
	hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	hipMalloc((void**)&result, sizeof(int));
	hipMalloc((void**)&time, sizeof(clock_t));

	//cudaMemcpy 将产生的随机数复制到显卡内存中 
	//cudaMemcpyHostToDevice - 从内存复制到显卡内存
	//cudaMemcpyDeviceToHost - 从显卡内存复制到内存
	hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

	// 在CUDA 中执行函数 语法：函数名称<<<block 数目, thread 数目, shared memory 大小>>>(参数...);
	sumOfSquares <<<1, 1, 0 >>>(gpudata, result, time);


	/*把结果从显示芯片复制回主内存*/

	int sum;
	clock_t time_used;

	//cudaMemcpy 将结果从显存中复制回内存
	hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);

	//Free
	hipFree(gpudata);
	hipFree(result);
	hipFree(time);

	printf("GPUsum: %d. Time: %d\n", sum, time_used);

	sum = 0;

	clock_t start = clock();
	for (int i = 0; i < DATA_SIZE; i++) {
		sum += data[i] * data[i];
	}
	clock_t end = clock();

	printf("CPUsum: %d. Time: %d\n", sum, end - start);

	return 0;
}
